#include "hip/hip_runtime.h"
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <fstream>
#include <math.h>
#include <random>
#include <iomanip>
#include <chrono>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/functional.h>
#include <thrust/transform.h>
#include <thrust/shuffle.h>
#include <thrust/random.h>
#include <thrust/device_ptr.h>

using namespace std;
struct city {
    int id;
    double x;
    double y;
};

// Creates a vector from stdin and number of cities
vector<city> createVector(int n_cities) {
    vector<city> cities;
    for (int i = 0; i < n_cities; i++) {
        city new_city;
        new_city.id = i;
        cin >> new_city.x;
        cin >> new_city.y;
        cities.push_back(new_city);
    }
    return cities;
}

__host__ __device__ double euclidean(city ct1, city ct2) { 
    return sqrt(pow(ct1.x - ct2.x, 2) + pow(ct1.y - ct2.y, 2));
}

void print_path(vector<city> cities) {
    for (int i = 0; i < (int) cities.size(); i++) {
        cout << cities[i].id << " ";
    }
    cout << endl;
}

__host__ __device__ double calc_tour(vector<city> cities) {
    double tour;
    int last_index = ((int) cities.size()) - 1; 
    for (int i = 0; i < last_index; i++) {
        tour += euclidean(cities[i], cities[i+1]);
    }
    // We have to return to the initial city
    tour +=  euclidean(cities[last_index], cities[0]);
    return tour;
}

int isPathShorter(vector<city> path_prev, vector<city> path_current) {
    if (calc_tour(path_current) < calc_tour(path_prev)) return 1;
    else return 0;
}

void print_err_output(vector<city> path, double tour) {
    cerr << tour << " ";
    for (int i = 0; i < (int) path.size(); i++) cerr << path[i].id << " ";
    cerr << "\n";
}

void print_test_err(vector<city> path, double tour) {
    cerr << tour << " " << 0 << endl;
    for (int i = 0; i < (int) path.size(); i++)  cout << path[i].id << " ";
    cout << endl;
}

void print_output_new(vector<city> path, double tour) {
    cout << tour << " " << 0 << '\n';
    for (int i = 0; i < (int) path.size(); i++)  cout << path[i].id << " ";
    cout << endl;
}

struct random_search {
    city swapped;
    double best_tour_inner;
    thrust::device_vector<city, thrust::device_allocator<city>> all_paths;
    int n_cities;
    random_search(vector<city> all_paths_, int n_cities_) : all_paths(all_paths_), n_cities(n_cities_) {};
    __host__ __device__
    float operator() (const int &index)
    {
    }
};

void sequential_random_search(vector<city> cities, int n_cities) {
    std::default_random_engine rng(10);
    vector<city> best_tour = cities;

    int last_index = (int) cities.size() - 1;

    for (int i = 0; i < 10*n_cities; i++) {
        vector<city> best_tour_inner = cities;
        
        // Shuffling vector
        // From https://stackoverflow.com/questions/6926433/how-to-shuffle-a-stdvector
        std::shuffle(std::begin(best_tour_inner), std::end(best_tour_inner), rng);

        vector<city> swapped = best_tour_inner;

        // Swapping vector items one by one
        for (int i = 0; i < last_index; i++) {     
            swap(swapped[i], swapped[i+1]);
            if (isPathShorter(best_tour_inner, swapped)) best_tour_inner = swapped;
        }

        if (isPathShorter(best_tour, best_tour_inner)) best_tour = best_tour_inner;
        
        //print_output_new(best_tour, calc_tour(best_tour));

        // For some reason, tests work with different outputs (stderr with format > tour index1 index2 ...)
        //print_err_output(best_tour, calc_tour(best_tour));
        }
    }

int main(int argc, char *argv[]) {
    thrust::default_random_engine generator(10);

    int n_cities;
    cin >> n_cities;

    vector<city> cities[n_cities];
    vector<city> OutputIterator[10*n_cities];

    // Populating indexes
    int indexes[10*n_cities];
    thrust::sequence(indexes, indexes + 10 * n_cities);
    
    // Number of possibilities is 10*n_cities, just like usual
    long n_possibilities = 10 * n_cities;

    // Creating cities vector from input file
    thrust::host_vector<city> h_cities = createVector(n_cities);

    // Copying to GPU
    thrust::device_vector<city> d_cities = h_cities;

    // All paths
    thrust::device_vector<city> all_paths(n_cities * n_possibilities);

    for (int i = 0; i < n_possibilities; i++) {
        // Copying original vector
        thrust::device_vector<city> d_swapped = d_cities;

        // Shuffle happens inside the GPU
        thrust::shuffle(d_swapped.begin(), d_swapped.end(), generator);

        for (int j = 0; j < n_cities; j++) {
            all_paths[(n_cities * i) + j] = d_swapped[j];
        }
    }

    thrust::device_vector<int> outer_paths(10*n_cities);

    // Supposing the best path is the first one
    float min_dist = calc_tour(outer_paths[0]);

    float min_dist = thrust::reduce(outer_paths.begin(), outer_paths.end(), min_dist, thrust::minimum<float>());
    

    // auto begin_random = std::chrono::high_resolution_clock::now();
    // auto end_random = std::chrono::high_resolution_clock::now();
    // auto elapsed_random = std::chrono::duration_cast<std::chrono::nanoseconds>(end_random - begin_random);
    // cout << "Sequential time: " << elapsed_random.count() * 1e-9 << endl;   
}